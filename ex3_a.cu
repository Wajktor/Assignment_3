#include "hip/hip_runtime.h"

#include <hiprand/hiprand.h>

#include <random>
#include <iostream>
#include <math.h>

#include <iomanip>
#include <string>
#include <map>
#include <cstdlib>
#include <ctime>

#include <fstream>


#include <stdio.h>

//#define NUM_PARTICLES 1e7
#define NUM_ITERATIONS 5000
#define dt 1.0f


__host__ __device__ float3 operator+(const float3 &a, const float3 &b) {
	return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

__host__ __device__ float3 operator*(const float &a, const float3 &b) {
	return make_float3(a*b.x, a*b.y, a*b.z);
}

__host__ __device__ float3 operator*(const float3 &a, const float &b) {
	return make_float3(b*a.x, b*a.y, b*a.z);
}





struct Particle {
	float3 position;
	float3 velocity;
};

__global__ void timeStep(Particle *par, int TPB){

	int index  = blockIdx.x * TPB + threadIdx.x;

	par[index].velocity = par[index].velocity + (-1.0f * par[index].position * dt); 
	
	par[index].position = par[index].position + par[index].velocity * dt; 

}


void timeStep_cpu(Particle *par, int NUM_PARTICLES){


	for(int i = 0; i<NUM_PARTICLES; ++i){
		par[i].velocity = par[i].velocity + (-1.0f * (par[i].position) * dt); 
	}


	for(int i = 0; i<NUM_PARTICLES; ++i){
		par[i].position = par[i].position + (par[i].velocity) * dt; 
	}

}



int main(int argc, char **argv) {

	std::cout << "hello";

	int TPB = atoi(argv[1]);
	int NUM_PARTICLES = atoi(argv[2]);


	int block_sizes[] = {16, 32, 64, 128, 256};
	int num_particles[] = {10000, 100, 1000, 10000, 100000};

	int block_array_len = 1;
	int num_particles_len = 1;



	Particle *particles;
	Particle *d_particles;
	Particle *particlesCompare;
	

	std::ofstream myfile("data.txt");


	for(int i = 0; i < block_array_len; ++i){
		for(int j = 0; j < num_particles_len; ++j){

			std::cout << "aaaasdasdasd";

			//TPB = block_sizes[i];
			//NUM_PARTICLES = num_particles[j];


			std::cout << "TPB: " << TPB << ", num particles: " << NUM_PARTICLES <<  "\n";
			
			int BLOCKS = (NUM_PARTICLES + TPB - 1)/TPB;


			//Particle  *particles = (Particle *)calloc(NUM_PARTICLES, sizeof(Particle));

			particles = (Particle*)malloc(sizeof(Particle)*NUM_PARTICLES);
			particlesCompare = (Particle*)malloc(sizeof(Particle)*NUM_PARTICLES);

			for(int k = 0; k < NUM_PARTICLES; ++k){
				particles[k].position = make_float3((float)rand()/(float)(RAND_MAX)*5.0f, (float)rand()/(float)(RAND_MAX)*5.0f, (float)rand()/(float)(RAND_MAX)*5.0f);
				particles[k].velocity = make_float3((float)rand()/(float)(RAND_MAX)*5.0f, (float)rand()/(float)(RAND_MAX)*5.0f, (float)rand()/(float)(RAND_MAX)*5.0f);
			}

			// Kopiera partiklarna in i compare
			for (int k = 0; k < NUM_PARTICLES; ++k)
			{
				particlesCompare[k].position = particles[k].position;
				particlesCompare[k].velocity = particles[k].velocity;
			} 
		/*	// verifiera att kopieringen ovan fungerar
			printf("ParticlesCompare:\nPosition: x=%f, y=%f, z=%f\n", particlesCompare[0].position.x, particlesCompare[0].position.y, particlesCompare[0].position.z);
			printf("Velocity: x=%f, y=%f, z=%f\n\n", particlesCompare[0].velocity.x, particlesCompare[0].velocity.y, particlesCompare[0].velocity.z);

			for (int i = 0; i < NUM_PARTICLES; ++i){
				printf("Position: x=%f, y=%f, z=%f\n", particles[i].position.x, particles[i].position.y, particles[i].position.z);
				printf("Velocity: x=%f, y=%f, z=%f\n\n", particles[i].velocity.x, particles[i].velocity.y, particles[i].velocity.z);
			}

		*/
			int size = sizeof(Particle)*NUM_PARTICLES;
			hipMalloc((void **)&d_particles, size);
				
			hipMemcpy(d_particles, particles, size, hipMemcpyHostToDevice);

			printf("Calculating on GPU... \n");


			// MAIN GPU LOOP
			clock_t begin = clock();
			for(int k = 0; k < NUM_ITERATIONS; ++k){
				hipMemcpy(d_particles, particles, size, hipMemcpyHostToDevice);
				timeStep <<< BLOCKS, TPB >>>(d_particles, TPB);
				hipMemcpy(particlesCompare, d_particles, size, hipMemcpyDeviceToHost);
			}




			clock_t end = clock();
			double time_elapsed = double(end-begin)/CLOCKS_PER_SEC;
			printf("Done, elapsed time: %f s\n", double(end-begin)/CLOCKS_PER_SEC );

				if (myfile.is_open()){
					myfile << "GPU " << TPB << " " << NUM_PARTICLES << " " << time_elapsed << "\n";
				}

			hipMemcpy(particlesCompare, d_particles, size, hipMemcpyDeviceToHost);
		/*
			printf("Particle values after timeStep:\n");
			for (int i = 0; i < NUM_PARTICLES; ++i){
				printf("Position: x=%f, y=%f, z=%f\n", particlesCompare[i].position.x, particlesCompare[i].position.y, particlesCompare[i].position.z);
				printf("Velocity: x=%f, y=%f, z=%f\n\n", particlesCompare[i].velocity.x, particlesCompare[i].velocity.y, particlesCompare[i].velocity.z);
			}
		*/

/*			printf("Calculating on CPU... \n");

			begin = clock();
			for(int k = 0; k < NUM_ITERATIONS; ++k){
				timeStep_cpu(particles, NUM_PARTICLES);
			}
			end = clock();

			time_elapsed = double(end-begin)/CLOCKS_PER_SEC;

			if (myfile.is_open()){
				myfile << "CPU 0 " << NUM_PARTICLES << " " << time_elapsed << "\n";
			}

			printf("Done, elapsed time: %f s\n", double(end-begin)/CLOCKS_PER_SEC ); */

		/*	for (int i = 0; i < NUM_PARTICLES; ++i){
				printf("Position: x=%f, y=%f, z=%f\n", particles[i].position.x, particles[i].position.y, particles[i].position.z);
				printf("Velocity: x=%f, y=%f, z=%f\n\n", particles[i].velocity.x, particles[i].velocity.y, particles[i].velocity.z);
			}
		*/

		}

	}

	myfile.close();

	return 0;
}
